#include "DeviceInfo.cuh"


void DeviceInfo::printCudaDeviceInfo(hipDeviceProp_t &prop) {
    printf("Device name:                                        %s\n", prop.name);
    printf("Global memory available on device:                  %zu\n", prop.totalGlobalMem);
    printf("Shared memory available per block:                  %zu\n", prop.sharedMemPerBlock);
    printf("Count of 32-bit registers available per block:      %i\n", prop.regsPerBlock);
    printf("Warp size in threads:                               %i\n", prop.warpSize);
    printf("Maximum pitch in bytes allowed by memory copies:    %zu\n", prop.memPitch);
    printf("Maximum number of threads per block:                %i\n", prop.maxThreadsPerBlock);
    printf("Maximum size of each dimension of a block[0]:       %i\n", prop.maxThreadsDim[0]);
    printf("Maximum size of each dimension of a block[1]:       %i\n", prop.maxThreadsDim[1]);
    printf("Maximum size of each dimension of a block[2]:       %i\n", prop.maxThreadsDim[2]);
    printf("Maximum size of each dimension of a grid[0]:        %i\n", prop.maxGridSize[0]);
    printf("Maximum size of each dimension of a grid[1]:        %i\n", prop.maxGridSize[1]);
    printf("Maximum size of each dimension of a grid[2]:        %i\n", prop.maxGridSize[2]);
    printf("Clock frequency in kilohertz:                       %i\n", prop.clockRate);
    printf("totalConstMem:                                      %zu\n", prop.totalConstMem);
    printf("Major compute capability:                           %i\n", prop.major);
    printf("Minor compute capability:                           %i\n", prop.minor);
    printf("Number of multiprocessors on device:                %i\n", prop.multiProcessorCount);
    printf("---------------------------------------------------------------\n");
}


void DeviceInfo::getCudaDeviceInfo() {
    int count;
    hipDeviceProp_t prop{};

    hipGetDeviceCount(&count);
    printf("Count CUDA devices = %i\n", count);

    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        printCudaDeviceInfo(prop);
    }
}

